#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Utilities and system includes
#include "DXTC.h"
#include "../elastic_kernel.h"


template <class T>
__device__ inline void swap(T &a, T &b)
{
    T tmp = a;
    a = b;
    b = tmp;
}

//__constant__ float3 kColorMetric = { 0.2126f, 0.7152f, 0.0722f };
__constant__ float3 kColorMetric = { 1.0f, 1.0f, 1.0f };

////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////
__device__ void sortColors(const float *values, int *ranks, cg::thread_group tile)
{
    const int tid = threadIdx.x;

    int rank = 0;

#pragma unroll

    for (int i = 0; i < 16; i++)
    {
        rank += (values[i] < values[tid]);
    }

    ranks[tid] = rank;

    cg::sync(tile);

    // Resolve elements with the same index.
    for (int i = 0; i < 15; i++)
    {
        if (tid > i && ranks[tid] == ranks[i])
        {
            ++ranks[tid];
        }
        cg::sync(tile);
    }
}

////////////////////////////////////////////////////////////////////////////////
// Load color block to shared mem
////////////////////////////////////////////////////////////////////////////////
__device__ void loadColorBlock(const uint *image, float3 colors[16], float3 sums[16], int xrefs[16], int blockOffset, cg::thread_block cta)
{
    const int bid = blockIdx.x + blockOffset;
    const int idx = threadIdx.x;

    __shared__ float dps[16];

    float3 tmp;

    cg::thread_group tile = cg::tiled_partition(cta, 16);

    if (idx < 16)
    {
        // Read color and copy to shared mem.
        uint c = image[(bid) * 16 + idx];

        colors[idx].x = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
        colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
        colors[idx].z = ((c >> 16) & 0xFF) * (1.0f / 255.0f);

        cg::sync(tile);
        // Sort colors along the best fit line.
        colorSums(colors, sums, tile);

        cg::sync(tile);

        float3 axis = bestFitLine(colors, sums[0], tile);

        cg::sync(tile);

        dps[idx] = dot(colors[idx], axis);

        cg::sync(tile);

        sortColors(dps, xrefs, tile);

        cg::sync(tile);

        tmp = colors[idx];

        cg::sync(tile);

        colors[xrefs[idx]] = tmp;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////
inline __device__ float3 roundAndExpand(float3 v, ushort *w)
{
    v.x = rintf(__saturatef(v.x) * 31.0f);
    v.y = rintf(__saturatef(v.y) * 63.0f);
    v.z = rintf(__saturatef(v.z) * 31.0f);

    *w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
    v.x *= 0.03227752766457f; // approximate integer bit expansion.
    v.y *= 0.01583151765563f;
    v.z *= 0.03227752766457f;
    return v;
}


__constant__ float alphaTable4[4] = { 9.0f, 0.0f, 6.0f, 3.0f };
__constant__ float alphaTable3[4] = { 4.0f, 0.0f, 2.0f, 2.0f };
__constant__ const int prods4[4] = { 0x090000,0x000900,0x040102,0x010402 };
__constant__ const int prods3[4] = { 0x040000,0x000400,0x040101,0x010401 };

#define USE_TABLES 1

////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
static __device__ float evalPermutation4(const float3 *colors, uint permutation, ushort *start, ushort *end, float3 color_sum)
{
    // Compute endpoints using least squares.
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable4[bits & 3] * colors[i];
        akku += prods4[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum = (9.0f * color_sum) - alphax_sum;
#else
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);

        if (bits & 2)
        {
            beta = (1 + beta) * (1.0f / 3.0f);
        }

        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    // alpha2, beta2, alphabeta and factor could be precomputed for each permutation, but it's faster to recompute them.
    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (0.111111111111f) * dot(e, kColorMetric);
}

static __device__ float evalPermutation3(const float3 *colors, uint permutation, ushort *start, ushort *end, float3 color_sum)
{
    // Compute endpoints using least squares.
#if USE_TABLES
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    int akku = 0;

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        alphax_sum += alphaTable3[bits & 3] * colors[i];
        akku += prods3[bits & 3];
    }

    float alpha2_sum = float(akku >> 16);
    float beta2_sum = float((akku >> 8) & 0xff);
    float alphabeta_sum = float((akku >> 0) & 0xff);
    float3 betax_sum = (4.0f * color_sum) - alphax_sum;
#else
    float alpha2_sum = 0.0f;
    float beta2_sum = 0.0f;
    float alphabeta_sum = 0.0f;
    float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

    // Compute alpha & beta for this permutation.
    for (int i = 0; i < 16; i++)
    {
        const uint bits = permutation >> (2*i);

        float beta = (bits & 1);

        if (bits & 2)
        {
            beta = 0.5f;
        }

        float alpha = 1.0f - beta;

        alpha2_sum += alpha * alpha;
        beta2_sum += beta * beta;
        alphabeta_sum += alpha * beta;
        alphax_sum += alpha * colors[i];
    }

    float3 betax_sum = color_sum - alphax_sum;
#endif

    const float factor = 1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

    float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
    float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

    // Round a, b to the closest 5-6-5 color and expand...
    a = roundAndExpand(a, start);
    b = roundAndExpand(b, end);

    // compute the error
    float3 e = a * a * alpha2_sum + b * b * beta2_sum + 2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

    return (0.25f) * dot(e, kColorMetric);
}

__device__ void evalAllPermutations(const float3 *colors, const uint *permutations, ushort &bestStart, ushort &bestEnd, uint &bestPermutation, float *errors, float3 color_sum, cg::thread_block cta)
{
    const int idx = threadIdx.x;

    float bestError = FLT_MAX;

    __shared__ uint s_permutations[160];

    for (int i = 0; i < 16; i++)
    {
        int pidx = idx + NUM_THREADS * i;

        if (pidx >= 992)
        {
            break;
        }

        ushort start, end;
        uint permutation = permutations[pidx];

        if (pidx < 160)
        {
            s_permutations[pidx] = permutation;
        }

        float error = evalPermutation4(colors, permutation, &start, &end, color_sum);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;
        }
    }

    if (bestStart < bestEnd)
    {
        swap(bestEnd, bestStart);
        bestPermutation ^= 0x55555555;    // Flip indices.
    }

    cg::sync(cta); // Sync here to ensure s_permutations is valid going forward

    for (int i = 0; i < 3; i++)
    {
        int pidx = idx + NUM_THREADS * i;

        if (pidx >= 160)
        {
            break;
        }

        ushort start, end;
        uint permutation = s_permutations[pidx];
        float error = evalPermutation3(colors, permutation, &start, &end, color_sum);

        if (error < bestError)
        {
            bestError = error;
            bestPermutation = permutation;
            bestStart = start;
            bestEnd = end;

            if (bestStart > bestEnd)
            {
                swap(bestEnd, bestStart);
                bestPermutation ^= (~bestPermutation >> 1) & 0x55555555;    // Flip indices.
            }
        }
    }

    errors[idx] = bestError;
}

////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ int findMinError(float *errors, cg::thread_block cta)
{
    const int idx = threadIdx.x;
    __shared__ int indices[NUM_THREADS];
    indices[idx] = idx;

    cg::sync(cta);

    for (int d = NUM_THREADS/2; d > 0; d >>= 1)
    {
        float err0 = errors[idx];
        float err1 = (idx + d) < NUM_THREADS ? errors[idx + d] : FLT_MAX;
        int index1 = (idx + d) < NUM_THREADS ? indices[idx + d] : 0;

        cg::sync(cta);

        if (err1 < err0)
        {
            errors[idx] = err1;
            indices[idx] = index1;
        }

        cg::sync(cta);
    }

    return indices[0];
}

////////////////////////////////////////////////////////////////////////////////
// Save DXT block
////////////////////////////////////////////////////////////////////////////////
__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation, int xrefs[16], uint2 *result, int blockOffset)
{
    const int bid = blockIdx.x + blockOffset;

    if (start == end)
    {
        permutation = 0;
    }

    // Reorder permutation.
    uint indices = 0;

    for (int i = 0; i < 16; i++)
    {
        int ref = xrefs[i];
        indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
    }

    // Write endpoints.
    result[bid].x = (end << 16) | start;

    // Write palette indices.
    result[bid].y = indices;
}

////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////
__launch_bounds__( 256 , 8 )
__global__ void compress(const uint *permutations, const uint *image, uint2 *result, int blockOffset, int *zc_slc)
{
	if (threadIdx.x == 0 && threadIdx.y == 0) atomicAdd(zc_slc, 1);

    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();

    const int idx = threadIdx.x;

    __shared__ float3 colors[16];
    __shared__ float3 sums[16];
    __shared__ int xrefs[16];

    loadColorBlock(image, colors, sums, xrefs, blockOffset, cta);

    cg::sync(cta);

    ushort bestStart, bestEnd;
    uint bestPermutation;

    __shared__ float errors[NUM_THREADS];

    evalAllPermutations(colors, permutations, bestStart, bestEnd, bestPermutation, errors, sums[0], cta);

    // Use a parallel reduction to find minimum error.
    const int minIdx = findMinError(errors, cta);

    cg::sync(cta);

    // Only write the result of the winner thread.
    if (idx == minIdx)
    {
        saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result, blockOffset);
    }
}

// Helper structs and functions to validate the output of the compressor.
// We cannot simply do a bitwise compare, because different compilers produce different
// results for different targets due to floating point arithmetic.

union Color32
{
    struct
    {
        unsigned char b, g, r, a;
    };
    unsigned int u;
};

union Color16
{
    struct
    {
        unsigned short b : 5;
        unsigned short g : 6;
        unsigned short r : 5;
    };
    unsigned short u;
};

struct BlockDXT1
{
    Color16 col0;
    Color16 col1;
    union
    {
        unsigned char row[4];
        unsigned int indices;
    };

    void decompress(Color32 colors[16]) const;
};

void BlockDXT1::decompress(Color32 *colors) const
{
    Color32 palette[4];

    // Does bit expansion before interpolation.
    palette[0].b = (col0.b << 3) | (col0.b >> 2);
    palette[0].g = (col0.g << 2) | (col0.g >> 4);
    palette[0].r = (col0.r << 3) | (col0.r >> 2);
    palette[0].a = 0xFF;

    palette[1].r = (col1.r << 3) | (col1.r >> 2);
    palette[1].g = (col1.g << 2) | (col1.g >> 4);
    palette[1].b = (col1.b << 3) | (col1.b >> 2);
    palette[1].a = 0xFF;

    if (col0.u > col1.u)
    {
        // Four-color block: derive the other two colors.
        palette[2].r = (2 * palette[0].r + palette[1].r) / 3;
        palette[2].g = (2 * palette[0].g + palette[1].g) / 3;
        palette[2].b = (2 * palette[0].b + palette[1].b) / 3;
        palette[2].a = 0xFF;

        palette[3].r = (2 * palette[1].r + palette[0].r) / 3;
        palette[3].g = (2 * palette[1].g + palette[0].g) / 3;
        palette[3].b = (2 * palette[1].b + palette[0].b) / 3;
        palette[3].a = 0xFF;
    }
    else
    {
        // Three-color block: derive the other color.
        palette[2].r = (palette[0].r + palette[1].r) / 2;
        palette[2].g = (palette[0].g + palette[1].g) / 2;
        palette[2].b = (palette[0].b + palette[1].b) / 2;
        palette[2].a = 0xFF;

        palette[3].r = 0x00;
        palette[3].g = 0x00;
        palette[3].b = 0x00;
        palette[3].a = 0x00;
    }

    for (int i = 0; i < 16; i++)
    {
        colors[i] = palette[(indices >> (2*i)) & 0x3];
    }
}

static int compareColors(const Color32 *b0, const Color32 *b1)
{
    int sum = 0;

    for (int i = 0; i < 16; i++)
    {
        int r = (b0[i].r - b1[i].r);
        int g = (b0[i].g - b1[i].g);
        int b = (b0[i].b - b1[i].b);
        sum += r*r + g*g + b*b;
    }

    return sum;
}

static int compareBlock(const BlockDXT1 *b0, const BlockDXT1 *b1)
{
    Color32 colors0[16];
    Color32 colors1[16];

    if (memcmp(b0, b1, sizeof(BlockDXT1)) == 0)
    {
        return 0;
    }
    else
    {
        b0->decompress(colors0);
        b1->decompress(colors1);

        return compareColors(colors0, colors1);
    }
}

int DXTC_start_mallocs(void *arg) {
    t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_DXTC_params *params = (t_DXTC_params *)kstub->params;

    if (params->printInfo) printf("%s Starting...\n\n", params->sSDKsample);

    // Load input image.
    unsigned char *data = NULL;
    uint W, H;

    char zero = '\0';
    params->image_path = sdkFindFilePath(INPUT_IMAGE, &zero);

    if (params->image_path == 0)
    {
        printf("Error, unable to find source image  <%s>\n", params->image_path);
        exit(EXIT_FAILURE);
    }

    if (!sdkLoadPPM4ub(params->image_path, &data, &W, &H))
    {
        printf("Error, unable to open source image file <%s>\n", params->image_path);

        exit(EXIT_FAILURE);
    }

    params->w = W;
    params->h = H;

    if (params->printInfo) printf("Image Loaded '%s', %d x %d pixels\n\n", params->image_path, params->w, params->h);

    // Allocate input image.
    params->memSize = params->w * params->h * 4;
    assert(0 != params->memSize);
    params->block_image = (uint *)malloc(params->memSize);

    // Convert linear image to block linear.
    for (uint by = 0; by < H/4; by++)
    {
        for (uint bx = 0; bx < W/4; bx++)
        {
            for (int i = 0; i < 16; i++)
            {
                const int x = i & 3;
                const int y = i / 4;
                params->block_image[(by * W/4 + bx) * 16 + i] =
                    ((uint *)data)[(by * 4 + y) * 4 * (W/4) + bx * 4 + x];
            }
        }
    }

    // copy into global mem
    params->d_data = NULL;
    checkCudaErrors(hipMalloc((void **) &params->d_data, params->memSize));

    //zc_slc malloc
	hipMalloc((void **)&params->zc_slc, sizeof(int));

    // Result
    params->d_result = NULL;
    params->compressedSize = (params->w / 4) * (params->h / 4) * 8;
    checkCudaErrors(hipMalloc((void **)&params->d_result, params->compressedSize));
    params->h_result = (uint *)malloc(params->compressedSize);

    // Compute permutations.
    computePermutations(params->permutations);
    return 0;
}

int DXTC_start_transfers(void *arg) {
    t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_DXTC_params *params = (t_DXTC_params *)kstub->params;

    // Copy permutations host to devie.
    params->d_permutations = NULL;
    checkCudaErrors(hipMalloc((void **) &params->d_permutations, 1024 * sizeof(uint)));
    checkCudaErrors(hipMemcpy(params->d_permutations, params->permutations, 1024 * sizeof(uint),
                               hipMemcpyHostToDevice));

    // Copy image from host to device
    checkCudaErrors(hipMemcpy(params->d_data, params->block_image, params->memSize, hipMemcpyHostToDevice));

    // Determine launch configuration and run timed computation numIterations times
    kstub->kconf.gridsize.x = ((params->w + 3) / 4) * ((params->h + 3) / 4); // rounds up by 1 block in each dim if %4 != 0
    kstub->total_tasks = kstub->kconf.gridsize.x; // rounds up by 1 block in each dim if %4 != 0

    hipDeviceProp_t deviceProp;

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDevice(&kstub->deviceId));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, kstub->deviceId));

    if (params->printInfo) {
        printf("Running DXT Compression on %u x %u image...\n", params->w, params->h);
        printf("\n%u Blocks, %u Threads per Block, %u Threads in Grid...\n\n",
            kstub->kconf.gridsize.x, NUM_THREADS, kstub->kconf.gridsize.x * NUM_THREADS);
    }

    return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int launch_orig_DXTC(void *arg)
{
    t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_DXTC_params *params = (t_DXTC_params *)kstub->params;

    //checkCudaErrors(hipDeviceSynchronize());

    compress<<<kstub->kconf.gridsize.x, kstub->kconf.blocksize.x>>>(params->d_permutations, params->d_data, (uint2 *)params->d_result, 0, params->zc_slc);

    getLastCudaError("compress");

    return 0;
}

int launch_slc_DXTC(void *arg)
{
    t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_DXTC_params *params = (t_DXTC_params *)kstub->params;

    //printf("Launch...\n");
    //checkCudaErrors(hipDeviceSynchronize());

    compress<<<kstub->total_tasks, kstub->kconf.blocksize.x, 0, *(kstub->execution_s)>>>(params->d_permutations, params->d_data, (uint2 *)params->d_result, kstub->kconf.initial_blockID, params->zc_slc);

    getLastCudaError("compress");

    return 0;
}

int DXTC_end_kernel(void *arg) {
    t_kernel_stub *kstub = (t_kernel_stub *)arg;
	t_DXTC_params *params = (t_DXTC_params *)kstub->params;

    // sync to host, stop timer, record perf
    checkCudaErrors(hipDeviceSynchronize());

    // copy result data from device to host
    checkCudaErrors(hipMemcpy(params->h_result, params->d_result, params->compressedSize, hipMemcpyDeviceToHost));

    // Write out result data to DDS file
    char output_filename[1024];
    strcpy(output_filename, params->image_path);
    strcpy(output_filename + strlen(params->image_path) - 3, "dds");
    FILE *fp = fopen(output_filename, "wb");

    if (fp == 0)
    {
        printf("Error, unable to open output image <%s>\n", output_filename);
        exit(EXIT_FAILURE);
    }

    DDSHeader header;
    header.fourcc = FOURCC_DDS;
    header.size = 124;
    header.flags  = (DDSD_WIDTH|DDSD_HEIGHT|DDSD_CAPS|DDSD_PIXELFORMAT|DDSD_LINEARSIZE);
    header.height = params->h;
    header.width = params->w;
    header.pitch = params->compressedSize;
    header.depth = 0;
    header.mipmapcount = 0;
    memset(header.reserved, 0, sizeof(header.reserved));
    header.pf.size = 32;
    header.pf.flags = DDPF_FOURCC;
    header.pf.fourcc = FOURCC_DXT1;
    header.pf.bitcount = 0;
    header.pf.rmask = 0;
    header.pf.gmask = 0;
    header.pf.bmask = 0;
    header.pf.amask = 0;
    header.caps.caps1 = DDSCAPS_TEXTURE;
    header.caps.caps2 = 0;
    header.caps.caps3 = 0;
    header.caps.caps4 = 0;
    header.notused = 0;
    fwrite(&header, sizeof(DDSHeader), 1, fp);
    fwrite(params->h_result, params->compressedSize, 1, fp);
    fclose(fp);

    // Make sure the generated image is correct.
    char zero = '\0';
    const char *reference_image_path = sdkFindFilePath(REFERENCE_IMAGE, &zero);

    if (reference_image_path == 0)
    {
        printf("Error, unable to find reference image\n");

        exit(EXIT_FAILURE);
    }

    fp = fopen(reference_image_path, "rb");

    if (fp == 0)
    {
        printf("Error, unable to open reference image\n");

        exit(EXIT_FAILURE);
    }

    fseek(fp, sizeof(DDSHeader), SEEK_SET);
    uint referenceSize = (params->w / 4) * (params->h / 4) * 8;
    uint *reference = (uint *)malloc(referenceSize);
    fread(reference, referenceSize, 1, fp);
    fclose(fp);

    printf("\nChecking accuracy...\n");
    float rms = 0;

    for (uint y = 0; y < params->h; y += 4)
    {
        for (uint x = 0; x < params->w; x += 4)
        {
            uint referenceBlockIdx = ((y/4) * (params->w/4) + (x/4));
            uint resultBlockIdx = ((y/4) * (params->w/4) + (x/4));

            int cmp = compareBlock(((BlockDXT1 *)params->h_result) + resultBlockIdx, ((BlockDXT1 *)reference) + referenceBlockIdx);

            if (cmp != 0.0f)
            {
                printf("Deviation at (%4d,%4d):\t%f rms\n", x/4, y/4, float(cmp)/16/3);
            }

            rms += cmp;
        }
    }

    rms /= params->w * params->h * 3;

    // Free allocated resources and exit
    checkCudaErrors(hipFree(params->d_permutations));
    checkCudaErrors(hipFree(params->d_data));
    checkCudaErrors(hipFree(params->d_result));
    free(params->image_path);
    // free(data); // Maybe free it right after we use it on mallocs phase
    free(params->block_image);
    free(params->h_result);
    // free(reference);

    printf("RMS(reference, result) = %f\n\n", rms);
    printf(rms <= ERROR_THRESHOLD ? "Test passed\n" : "Test failed!\n");
    /* Return zero if test passed, one otherwise */
    return rms > ERROR_THRESHOLD;

    printf("...end\n");

    return 0;
}
